#include "hip/hip_runtime.h"
#include "globalFun.h"
#include <cstdio>

__global__ void helloWorld() {
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from CPU!\n");
    helloWorld<<<1, 10>>>();
    CHECK(hipDeviceSynchronize());
    hipDeviceReset();
    return 0;
}
